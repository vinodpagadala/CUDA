#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include "cpu_bitmap.h"
#include "bitmap_help.h"

#define CHANNELS 4
__global__ void rgbToGray(unsigned char *grayImage, int width, int height) 
{ 
	
		int rgbOffset= (blockIdx.x + blockIdx.y * gridDim.x) * CHANNELS;
		printf("rgbOffset %d",rgbOffset);	
		unsigned char red = grayImage[rgbOffset];
		unsigned char green = grayImage[rgbOffset + 1];
		unsigned char blue = grayImage[rgbOffset + 2];
		int grayOffset= 0.21f*red + 0.71f*green + 0.07f*blue;
		
		grayImage[rgbOffset] = grayImage[rgbOffset + 1] = grayImage[rgbOffset + 2]=  grayOffset;
		__syncthreads();
		
}

__host__ void imgProc(unsigned char *map, int size, int width, int height) {
	
	unsigned char* grayImage;
    size_t imgSize = size;

   hipMalloc((void**)&grayImage,imgSize);
   hipMemcpy(grayImage,map,imgSize,hipMemcpyHostToDevice);

   const dim3 gridSize(width,height,1);
   const dim3 blockSize(1,1,1);
   rgbToGray<<<gridSize,blockSize>>>(grayImage, width, height);
   hipDeviceSynchronize();
   hipMemcpy(map,grayImage,imgSize,hipMemcpyDeviceToHost);
   return;
}

int main(void) {
   char fname[50];
   FILE* infile;
   unsigned short ftype;
   tagBMFH bitHead;
   tagBMIH bitInfoHead;
   tagRGBQ *pRgb;

   printf("Please enter the .bmp file name: ");
   scanf("%s", fname);
   strcat(fname,".bmp");
   infile = fopen(fname, "rb");

   if (infile != NULL) {
      printf("File open successful.\n");
      fread(&ftype, 1, sizeof(unsigned short), infile);
      if (ftype != 0x4d42)
      {
         printf("File not .bmp format.\n");
         return 1;
      }
      fread(&bitHead, 1, sizeof(tagBMFH), infile);
      fread(&bitInfoHead, 1, sizeof(tagBMIH), infile);      
   }
   else {
      printf("File open fail.\n");
      return 1;
   }

   if (bitInfoHead.biBitCount < 24) {
      long nPlateNum = long(pow(2, double(bitInfoHead.biBitCount)));
      pRgb = (tagRGBQ *)malloc(nPlateNum * sizeof(tagRGBQ));
      memset(pRgb, 0, nPlateNum * sizeof(tagRGBQ));
      int num = fread(pRgb, 4, nPlateNum, infile);
   }

   int width = bitInfoHead.biWidth;
   int height = bitInfoHead.biHeight;
   int l_width = 4 * ((width * bitInfoHead.biBitCount + 31) / 32);
   long nData = height * l_width;
   unsigned char *pColorData = (unsigned char *)malloc(nData);
   memset(pColorData, 0, nData);
   fread(pColorData, 1, nData, infile);

   fclose(infile);
   
   CPUBitmap dataOfBmp(width, height);
   unsigned char *map = dataOfBmp.get_ptr();

   if (bitInfoHead.biBitCount < 24) {
      int k, index = 0;
      if (bitInfoHead.biBitCount == 1) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 8;
               mixIndex = pColorData[k];
               if (j % 8 < 7) mixIndex = mixIndex << (7 - (j % 8));
               mixIndex = mixIndex >> 7;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 2) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 4;
               mixIndex = pColorData[k];
               if (j % 4 < 3) mixIndex = mixIndex << (6 - 2 * (j % 4));
               mixIndex = mixIndex >> 6;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 4) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 2;
               mixIndex = pColorData[k];
               if (j % 2 == 0) mixIndex = mixIndex << 4;
               mixIndex = mixIndex >> 4;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 8) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j;
               mixIndex = pColorData[k];
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 16) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j * 2;
               unsigned char shortTemp = pColorData[k + 1] << 8;
               mixIndex = pColorData[k] + shortTemp;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
   }
   else {
      int k, index = 0;
      for (int i = 0; i < height; i++)
         for (int j = 0; j < width; j++) {
            k = i * l_width + j * 3;
            map[index * 4 + 0] = pColorData[k + 2];
            map[index * 4 + 1] = pColorData[k + 1];
            map[index * 4 + 2] = pColorData[k];
            index++;
         }
   }

   imgProc(map, dataOfBmp.image_size(), width, height);
   dataOfBmp.display_and_exit();
   return 0;
}
