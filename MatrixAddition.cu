
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
       }


int main(){

int A[N][N] = {{10,5},{8,5}};
int B[N][N] = {{2,3},{9,6}};
int C[N][N] = {{0,0},{0,0}};    

int (*pA)[N], (*pB)[N], (*pC)[N];

hipMalloc((void**)&pA, (N*N)*sizeof(int));
hipMalloc((void**)&pB, (N*N)*sizeof(int));
hipMalloc((void**)&pC, (N*N)*sizeof(int));

hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(pC, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

int numBlocks = 1;
dim3 threadsPerBlock(N,N);
MatAdd<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);

hipMemcpy(C, pC, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

int i, j; printf("C = \n");
for(i=0;i<N;i++){
    for(j=0;j<N;j++){
        printf("%d ", C[i][j]);
    }
    printf("\n");
}

hipFree(pA); 
hipFree(pB); 
hipFree(pC);

printf("\n");

return 0;
}
